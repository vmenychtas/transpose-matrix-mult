#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define TILE_WIDTH 16

void cudaErrorCheck() {
	hipError_t code = hipGetLastError();
	if (code != hipSuccess) {
		printf("[CUDA error]: %s\n", hipGetErrorString(code));
		exit(1);
	}
}

__global__ void matrixMultiply(double* mA, double* mC, int rows, int cols){
	__shared__ double sT[TILE_WIDTH][TILE_WIDTH];
	__shared__ double sN[TILE_WIDTH][TILE_WIDTH];
	double T, N;
	int k;

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row = blockIdx.y*blockDim.y + ty;
	int col = blockIdx.x*blockDim.x + tx;

	double val = 0.0;
	for (k = 0; k < gridDim.y; k++) {
		T = 0.0; N = 0.0;
		if ((row < cols) && (k*TILE_WIDTH + tx < rows)) {
			T = mA[(k*TILE_WIDTH + tx)*cols + row];
		}
		if ((col < cols) && (k*TILE_WIDTH + ty < rows)) {
			N = mA[(k*TILE_WIDTH + ty)*cols + col];
		}
		sT[ty][tx] = T;
		sN[ty][tx] = N;
		__syncthreads();

		if (row <= col) {
			val += sT[ty][15] * sN[15][tx];
			val += sT[ty][14] * sN[14][tx];
			val += sT[ty][13] * sN[13][tx];
			val += sT[ty][12] * sN[12][tx];
			val += sT[ty][11] * sN[11][tx];
			val += sT[ty][10] * sN[10][tx];
			val += sT[ty][9] * sN[9][tx];
			val += sT[ty][8] * sN[8][tx];
			val += sT[ty][7] * sN[7][tx];
			val += sT[ty][6] * sN[6][tx];
			val += sT[ty][5] * sN[5][tx];
			val += sT[ty][4] * sN[4][tx];
			val += sT[ty][3] * sN[3][tx];
			val += sT[ty][2] * sN[2][tx];
			val += sT[ty][1] * sN[1][tx];
			val += sT[ty][0] * sN[0][tx];
		}
		__syncthreads();
	}

	if ((row <= col) && (col < cols)) {
		mC[row*cols + col] = val;
		if (row != col) {
			mC[col*cols + row] = val;
		}
	}
}

int main(int argc, char *argv[]){
	int i, j, pr_dev = 0;

	if (argv[1] == NULL){
		printf("Matrix dimensions not specified.\n\n");
		exit(1);
	}	else if (argv[2] == NULL) {
		printf("Assuming square matrix.\n\n");
		argv[2] = argv[1];
	}	else if (strcmp(argv[2], "-pd") == 0) {
		printf("Assuming square matrix.\n\n");
		argv[2] = argv[1];
		pr_dev = 1;
	}	else if (argv[3] != NULL) {
		if (strcmp(argv[3], "-pd") == 0) {
			pr_dev = 1;
		}
	}
	int rows = atoi(argv[1]); // A transposed width
	int cols = atoi(argv[2]); // A transposed height -> Dimensions of result C
	/* Allocation and initialization of matrices on host -----------------------*/
	double *hA, *hC;

	hA = (double *)malloc(cols*rows*sizeof(double));
	hC = (double *)malloc(cols*cols*sizeof(double));
	srand(time(NULL));
	for (i = 0; i < rows; i++){
		for (j = 0; j < cols; j++){
			hA[i*cols + j] = rand() / 1000000.0;
		}
	}
	/* Allocation of matrices on the GPU ---------------------------------------*/
	double *dA, *dC;

	hipMalloc((void **)&dA, cols*rows*sizeof(double));
	hipMalloc((void **)&dC, cols*cols*sizeof(double));
	hipMemcpy(dA, hA, rows*cols*sizeof(double), hipMemcpyHostToDevice);
	/* Get device properties & print them if asked -----------------------------*/
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, 0);

	if (pr_dev == 1) {
		printf("   [ Device :: %s ]\n", props.name);
		printf("  Number of SMs ---------------: %d\n", props.multiProcessorCount);
		printf("  Global memory ---------------: %lu MB\n", props.totalGlobalMem / 1048576);
		printf("  Constant memory -------------: %lu KB\n", props.totalConstMem / 1024);
		printf("  Threads per warp ------------: %d\n", props.warpSize);
		printf("  Max threads per block -------: %d\n", props.maxThreadsPerBlock);
		printf("  Max registers per block -----: %dK\n", props.regsPerBlock / 1024);
		printf("  Max shared memory per block -: %lu KB\n", props.sharedMemPerBlock / 1024);
		printf("  Max block dimension ---------: %d x %d x %d\n", props.maxThreadsDim[0], props.maxThreadsDim[1], props.maxThreadsDim[2]);
		printf("  Max grid dimension ----------: %d x %d x %d\n\n", props.maxGridSize[0], props.maxGridSize[1], props.maxGridSize[2]);
	}
	/* Kernel invocation & computation timing ----------------------------------*/
	unsigned int block_width = TILE_WIDTH;
	unsigned int grid_dim = (cols - 1) / block_width + 1;
	dim3 dimGrid(grid_dim, grid_dim, 1);
	dim3 dimBlock(block_width, block_width, 1);

	printf("Matrix A ---: %d x %d\n", rows, cols);
	printf("Grid size --: %u x %u\n", grid_dim, grid_dim);
	printf("Block size -: %u x %u\n\n", block_width, block_width);

	hipEvent_t com_begin, com_end, mem_begin, mem_end;
	hipEventCreate(&com_begin); hipEventCreate(&com_end);
	hipEventCreate(&mem_begin); hipEventCreate(&mem_end);

	hipEventRecord(com_begin, 0);
	matrixMultiply<<<dimGrid , dimBlock>>>(dA, dC, rows, cols);
	hipEventRecord(com_end, 0);
	hipEventSynchronize(com_end);

	hipEventRecord(mem_begin, 0);
	hipMemcpy(hC, dC, cols*cols*sizeof(double), hipMemcpyDeviceToHost);
	hipEventRecord(mem_end, 0);
	hipEventSynchronize(mem_end);

	float com_time = 0;
	float mem_time = 0;
	hipEventElapsedTime(&com_time, com_begin, com_end);
	hipEventElapsedTime(&mem_time, mem_begin, mem_end);
	printf("Computation time: %.3f ms\n", com_time);
	printf("Memory transfer: %.3f ms\n", mem_time);
	/* Print output to file for computation evaluation -------------------------*/
	if (cols*rows <= 4096){
		FILE *f = fopen("data.txt", "w");
		if (f == NULL) {
			printf("Error: failed to open file.\n");
			exit(1);
		}

		for (i = 0; i < cols; i++){
			for (j = 0; j < cols; j++){
				fprintf(f, "%f ", hC[i*cols + j]);
			}
			fprintf(f, ";\n");
		}
	}

	free(hA);	hipFree(dA);
	free(hC);	hipFree(dC);
	cudaErrorCheck();
	return 0;
}
